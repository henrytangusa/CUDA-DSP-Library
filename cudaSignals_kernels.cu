
#include <hip/hip_runtime.h>
__global__ void SinWaveKernel(float *data, int size, 
		              float amp, float freq, float ip, float sr, float tt)
{

};
	
__global__ void CosWaveKernel(float *data, int size, 
		              float amp, float freq, float ip, float sr, float tt)
{

};



	
void cudaSinWave(float *data, int size, 
                   float amp, float freq, float ip, float sr, float tt)
{
      SinWaveKernel<<<1024, 1024>>>(data, size, amp, freq, ip, sr, tt);

};
	
void cudaCosWave(float *data, int size, 
                   float amp, float freq, float ip, float sr, float tt)
{
      CosWaveKernel<<<1024, 1024>>>(data, size, amp, freq, ip, sr, tt);
};
