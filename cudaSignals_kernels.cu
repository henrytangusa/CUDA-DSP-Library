#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"


__global__ void SinWaveKernel(float *data, int size, 
                float amp, float freq, float ip, int sr, float tt)
{

}
                              

__global__ void CosWaveKernel(float *data, int size, 
                float amp, float freq, float ip, int sr, float tt)
{

}
                              
void SinWaveKernel(float *data, int size, 
                   float amp, float freq, float ip, int sr, float tt)
{
        SinWaveKernel<<1024, BLOCK >> (data, size, amp, freq, ip, sr, tt);
        check_error(hipPeekAtLastError());
}


void CosWaveKernel(float *data, int size, 
                   float amp, float freq, float ip, int sr, float tt)
{
        SinWaveKernel<<1024, BLOCK >> (data, size, amp, freq, ip, sr, tt);
        check_error(hipPeekAtLastError());
}

