#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"
#include"hip/hip_runtime.h"


__global__ void runSinWaveKernel(float *data, int size, 
                float amp, float freq, float ip, int sr)  // tt time interval is not needed

{
     unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
     if (x >= size)  return;
     float sampleInterval = sr / freq;   // Sample rate/signal freq
     data[x] = amp * sinf((2.0 * HIP_PI_F * (float)x) / sampleInterval + ip);
}
                              

__global__ void runCosWaveKernel(float *data, int size, 
                float amp, float freq, float ip, int sr)
{
     unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
     if (x >= size)  return;
     float sampleInterval = sr / freq;   // Sample rate/signal freq
     data[x] = amp * sinf((2.0 * HIP_PI_F * (float)x) / sampleInterval + ip);
}
                              
bool SinWaveKernel(float *data, int size, 
                   float amp, float freq, float ip, int sr)
{
        runSinWaveKernel<<<1024, BLOCK >>> (data, size, amp, freq, ip, sr);
        hipDeviceSynchronize();
        return hipPeekAtLastError() == hipSuccess;
}


bool CosWaveKernel(float *data, int size, 
                   float amp, float freq, float ip, int sr, float tt)
{
        runCosWaveKernel<<<1024, BLOCK >>> (data, size, amp, freq, ip, sr);
        hipDeviceSynchronize();
        return hipPeekAtLastError() == hipSuccess;
}

